#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctype.h>
#include <cstdlib>
#include <fstream>
#include <vector>
#include <iostream>
#include <string>
#include <sstream>
#include <chrono>
#include <time.h>

#include "Image.h"
__device__ int medHelper(int* arr, int winSize, int height, int col, int row) {
	int g = 0;
	int rad = winSize / 2;
	int* mini = (int*)malloc(sizeof(int) * winSize * winSize);
	//printf("col: %d srow: %d h-rad: %d rad: %d \n", col, sRow, (height - rad), rad);
	if (col >= rad && col < (height - rad) && row >= rad && row < (height - rad)) {
		for (int k = col - rad; k <= col + rad; k++) {
			for (int j = row - rad; j <= row + rad; j++) {
				if (((j * height) + k) > (512 * 512)) {
					printf("error here \n");
					return;
				}
				///printf("col: %d srow: %d k: %d j: %d pos: %d \n", col, sRow, k, j, (j * height) + k);
				//printf("at %d arr: %d g: %d \n", (j * height) + k, arr[(j * height) + k], g);
				mini[g] = arr[(j * height) + k];
				g++;
			}
		}
		int key;
		int key2;
		for (int i = 1; i < (winSize * winSize); i++) {
			key = mini[i];
			key2 = i - 1;
			while (key2 >= 0 && mini[key2] > key) {
				mini[key2 + 1] = mini[key2];
				key2 = key2 - 1;
			}
			mini[key2 + 1] = key;
		}
		int median = mini[(winSize * winSize) / 2];//4
		//free(mini);
		return median;
	}
	return 0;
}
__global__ void medianFilter(int* arr, int* copy, int winSize, int height, int width, int config) {
	//same thing but 1d array
	int row = 0;
	int col = 0;
	int rad = winSize / 2;
		//rad: can be 1, 3, 5, or 7
		if (blockIdx.x == 0 && threadIdx.x == 0) {
			for (int k = 0; k < rad; k++) {
				for (int i = 0; i < height; i++) {
					//copy border to handle edge case
					copy[i + (k * height)] = arr[i + (k * height)];
					copy[k + (i * height)] = arr[k + (i * height)];
					copy[(height - 1 - k) + (i * height)] = arr[(height - 1 - k) + (i * height)];
					copy[i + ((height - 1 - k) * height)] = arr[i + ((height - 1 - k) * height)];
				}
			}
		}
		
		int start;
		if (config == 1) {
			start = ((blockIdx.x + 1) * 256) + (threadIdx.x * 4096) - 256;
		}
		else if (config == 2) {
			start = (blockIdx.x * 4096) + ((threadIdx.x + 1) * 256) - 256;
		}
		else if (config == 3) {
			start = (blockIdx.x * 2048) + ((threadIdx.x + 1) * 256) - 256; //8 threads, 128 blocks
		}
		else if (config == 4) {
			start = ((blockIdx.x + 1) * 256) + (threadIdx.x * 2048) - 256; //128 threads, 8 blocks
		}
		int end = start + 256;
		int sRow = start / 512;
		int sCol = start % 512;
		int sc = sCol + 256;
		//printf("start: %d rad: %d sCol: %d sRow: %d \n", start, rad, sCol, sRow);
		int* mini = (int*)malloc(sizeof(int) * winSize * winSize);
		//int mini[225];
		for (col = sCol; col < sc; col++) {
			int g = 0;
			//printf("col: %d srow: %d h-rad: %d rad: %d \n", col, sRow, (height - rad), rad);
			if (col >= rad && col < (height - rad) && sRow >= rad && sRow < (height - rad)) {
				for (int k = col - rad; k <= col + rad; k++) {
					for (int j = sRow - rad; j <= sRow + rad; j++) {
						if (((j * height) + k) > (512 * 512)) {
							printf("error here \n");
							return;
						}
						///printf("col: %d srow: %d k: %d j: %d pos: %d \n", col, sRow, k, j, (j * height) + k);
						//printf("at %d arr: %d g: %d \n", (j * height) + k, arr[(j * height) + k], g);
						mini[g] = arr[(j * height) + k];
						g++;
					}
				}
				int key;
				int key2;
				for (int i = 1; i < (winSize * winSize); i++) {
					key = mini[i];
					key2 = i - 1;
					while (key2 >= 0 && mini[key2] > key) {
						mini[key2 + 1] = mini[key2];
						key2 = key2 - 1;
					}
					mini[key2 + 1] = key;
				}
				int median = mini[(winSize * winSize) / 2];//4
				//printf("c: %d r: %d med: %d \n", col, sRow, median);
				
				if (((sRow * height) + col) >= (512 * 512)) {
					printf("prob \n");
					return;
				}
				//int median = medHelper(arr, winSize, height, col, row);
				copy[(sRow * height) + col] = median;
			}
		}
		free(mini);	
}
using namespace std;
int readImage(char fname[], Image& image);
int readImageHeader(char fname[], int& N, int& M, int& Q, bool& type);
int writeImage(char fname[], Image& image);
int main(int argc, char** argv) {
	if (argc != 4) {
		cout << "error: needs to be argc 3, format should be <int filter> <input file> <output file> \n";
		cout << "argc: " << argc << "\n";
		return 0;
	}
	int filterSize = atoi(argv[1]);
	if (filterSize != 3 && filterSize != 7 && filterSize != 11 && filterSize != 15) {
		cout << "error: filter size must be 3, 7, 11 or 15 \n";
		return 0;
	}
	string inputFile = argv[2];
	string outputFile = argv[3];
	int col = 0;
	int row = 0;
	int M, N, Q; // rows, cols, grayscale
	bool type;
	// read image header
	readImageHeader(argv[2], N, M, Q, type);
	cout << "header: N " << N << " M " << M << " Q " << Q << "\n";
	Image image(N, M, Q);
	readImage(argv[2], image);
	int configVer = 0;
	//make 1d array with cudamalloc
	//convert 2d array to 1d array
	int* bigOne = (int*) malloc(sizeof(int) * N * M);
	//size_t pitch;
	//try converting to 1d array first
	int z = 0;
	int* gold1 = (int*)malloc(sizeof(int) * N * M);
	int* gold2 = (int*)malloc(sizeof(int) * N * M);
	int* neutral = (int*)malloc(sizeof(int) * N * M);
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			*(bigOne+z) = image.getPixelVal(j, i);
			*(gold1 + z) = image.getPixelVal(j, i);
			*(neutral + z) = image.getPixelVal(j, i);
			z++;
		}
	}
	//test
	int* bigDev;
	hipMalloc(&bigDev, (sizeof(int) * M * N));
	hipMemcpy(bigDev, bigOne, (sizeof(int) * M * N), hipMemcpyHostToDevice);
	int* bigDev2;
	//hipError_t cudStat;
	hipMalloc(&bigDev2, sizeof(int) * M * N);
	//my gpu has maxwell architecture, meaning that its compute cabability is 5.2 and onwards, can run at most 1024 threads per block and approx 16 active blocks

	int rad = filterSize / 2;
	int* cp = (int*)malloc(sizeof(int) * N * M);
	int threadsPerBlock = 1024;
	//int blocksPerGrid = (512 * 512 + threadsPerBlock - 1) / threadsPerBlock;
	int* mini = (int*)malloc(sizeof(int) * filterSize * filterSize);
	//dim3 threadBlocks = (64, 64);
	//dim3 blockTot = (16, 16); //each block has 256 to deal with
	dim3 tb1 = (64, 64);
	dim3 bt1 = (16, 16);
	dim3 tb2 = (16, 16);
	dim3 bt2 = (64, 64);
	dim3 tb3 = (8, 8);
	dim3 bt3 = (128, 128);
	dim3 tb4 = (128, 128);
	dim3 bt4 = (8, 8);
	clock_t s1 = clock();
	//pre
	/*for (int k = 0; k < rad; k++) {
		for (int i = 0; i < M; i++) {
			//copy border to handle edge case
			//copy[i + (k * height)] = arr[i + (k * height)];
			hipMemcpy((bigDev2+ i + (k * N)), (bigDev + i + (k * N)), sizeof(int), hipMemcpyDeviceToDevice);
			//copy[k + (i * height)] = arr[k + (i * height)];
			hipMemcpy((bigDev2 + k + (i * N)), (bigDev + k + (i * N)), sizeof(int), hipMemcpyDeviceToDevice);
			//copy[(height - 1 - k) + (i * height)] = arr[(height - 1 - k) + (i * height)];
			hipMemcpy((bigDev2 + (N - 1 - k) + (i * N)), (bigDev + (N - 1 - k) + (i * N)), sizeof(int), hipMemcpyDeviceToDevice);
			//copy[i + ((height - 1 - k) * height)] = arr[i + ((height - 1 - k) * height)];
			hipMemcpy((bigDev2 + i + (N - 1 - k) * N), (bigDev + i + (N - 1 - k) * N), sizeof(int), hipMemcpyDeviceToDevice);
		}
	}*/
	//hipError_t err2 = hipDeviceGetLimit();
	//hipError_t err2 = hipDeviceSetLimit(hipLimitMallocHeapSize, 264217728);
	//cout << "err2: " << hipGetErrorString(err2) << "\n";
	hipError_t err;
	medianFilter << <bt1, tb1 >> > (bigDev, bigDev2, filterSize, M, N, 1);
	err = hipDeviceSynchronize();
	cout << "err " << hipGetErrorString(err) << "\n";
	hipMemcpy(bigOne, bigDev2, sizeof(int) * N * M, hipMemcpyDeviceToHost);
	clock_t e1 = clock();
	clock_t s2 = clock();
	medianFilter << <bt2, tb2 >> > (bigDev, bigDev2, filterSize, M, N, 2);
	err = hipDeviceSynchronize();
	cout << "err " << hipGetErrorString(err) << "\n";
	hipMemcpy(bigOne, bigDev2, sizeof(int) * N * M, hipMemcpyDeviceToHost);
	clock_t e2 = clock();
	clock_t s3 = clock();
	medianFilter << <bt3, tb3 >> > (bigDev, bigDev2, filterSize, M, N, 3);
	err = hipDeviceSynchronize();
	cout << "err " << hipGetErrorString(err) << "\n";
	hipMemcpy(bigOne, bigDev2, sizeof(int) * N * M, hipMemcpyDeviceToHost);
	clock_t e3 = clock();
	clock_t s4 = clock();
	medianFilter << <bt4, tb4 >> > (bigDev, bigDev2, filterSize, M, N, 4);
	err = hipDeviceSynchronize();
	cout << "err " << hipGetErrorString(err) << "\n";
	hipMemcpy(bigOne, bigDev2, sizeof(int) * N * M, hipMemcpyDeviceToHost);
	clock_t e4 = clock();

	for (int k = 0; k < rad; k++) {
		for (int i = 0; i < M; i++) {
			//copy border
			gold2[i + (k * M)] = gold1[i + (k * M)];
			gold2[k + (i * M)] = gold1[k + (i * M)];
			gold2[(M - 1 - k) + (i * M)] = gold1[(N - 1 - k) + (i * M)];
			gold2[i + ((M - 1 - k) * M)] = gold1[i + ((N - 1 - k) * M)];
		}
	}
	for (row = rad; row < M - rad; row++) {
		for (col = rad; col < N - rad; col++) {
			int g = 0;
			for (int k = row - rad; k <= row + rad; k++) {
				for (int j = col - rad; j <= col + rad; j++) {
					mini[g] = gold1[k + (j * N)];
					g++;
				}
			}
			//sort mini via insertion sort
			int key;
			int key2;
			for (int i = 1; i < (filterSize * filterSize); i++) {
				key = mini[i];
				key2 = i - 1;
				while (key2 >= 0 && mini[key2] > key) {
					mini[key2 + 1] = mini[key2];
					key2 = key2 - 1;
				}
				mini[key2 + 1] = key;
			}
			//the resulting array length of the window will always be odd so can just take size/2 to be the median after sort
			int median = mini[(filterSize * filterSize) / 2];//4
			gold2[row + (col * M)] = median;
		}
	}
	free(mini);
	Image cp2(N, M, Q);
	for (int i = 0; i < M; i++) {
		for (int k = 0; k < N; k++) {
			cp2.setPixelVal(i, k, bigOne[i + (N * k)]);
		}
	}
	writeImage(argv[3], cp2);
	cout << "filter " << filterSize << " N " << N << " M " << M << " rad " << rad << "\n";
	float correctC = 0;
	float totC = N * M;
	for (int i = 0; i < M; i++) {
		for (int k = 0; k < N ; k++) {
			//cout << "at " << (i + (k * N)) << " b " << bigOne[i + (k * N)] << " g " << gold2[i + (k * N)] << " n " << neutral[i + (k * N)] << "\n";
			if (bigOne[i + (k * N)] == gold2[i + (k * N)]) {
				correctC++;
			}
			else {
				//cout << "at " << (i + (k * N)) << " b " << bigOne[i + (k * N)] << " g " << gold2[i + (k * N)] << " n " << neutral[i + (k * N)] << "\n";
			}
		}
	}
	float percent = correctC / totC;
	cout << "percent correct (version 4): " << percent * 100 << "\n";
	double time1 = (double)(e1 - s1);
	time1 = time1 / CLOCKS_PER_SEC;
	double time2 = (double)(e2 - s2);
	time2 = time2 / CLOCKS_PER_SEC;
	double time3 = (double)(e3 - s3);
	time3 = time3 / CLOCKS_PER_SEC;
	double time4 = (double)(e4 - s4);
	time4 = time4 / CLOCKS_PER_SEC;
	cout << "note: each thread processes medians for all 4 versions \n";
	cout << "time for kernel to run with 64 blocks and 16 threads per block: " << time1 << " seconds \n";
	cout << "time for kernel to run with 16 blocks and 64 threads per block: " << time2 << " seconds \n";
	cout << "time for kernel to run with 8 blocks and 128 threads per block: " << time3 << " seconds \n";
	cout << "time for kernel to run with 128 blocks and 8 threads per block: " << time4 << " seconds \n";
	return 0;
}

int readImage(char fname[], Image& image)
{
	int i, j;
	int N, M, Q;
	unsigned char* charImage;
	char header[100], * ptr;
	ifstream ifp;

	ifp.open(fname, ios::in | ios::binary);

	if (!ifp)
	{
		cout << "Can't read image: " << fname << endl;
		exit(1);
	}

	// read header
	ifp.getline(header, 100, '\n');
	if ((header[0] != 80) || (header[1] != 53))
	{
		cout << "Image " << fname << " is not PGM" << endl;
		exit(1);
	}

	ifp.getline(header, 100, '\n');
	while (header[0] == '#')
		ifp.getline(header, 100, '\n');

	M = strtol(header, &ptr, 0);
	N = atoi(ptr);

	ifp.getline(header, 100, '\n');
	Q = strtol(header, &ptr, 0);

	charImage = (unsigned char*) new unsigned char[M * N];

	ifp.read(reinterpret_cast<char*>(charImage), (M * N) * sizeof(unsigned char));

	if (ifp.fail())
	{
		cout << "Image " << fname << " has wrong size" << endl;
		exit(1);
	}

	ifp.close();

	int val;
	for (i = 0; i < N; i++)
		for (j = 0; j < M; j++)
		{
			val = (int)charImage[i * M + j];
			image.setPixelVal(i, j, val);
		}

	delete[] charImage;
	return (1);
}
int readImageHeader(char fname[], int& N, int& M, int& Q, bool& type)
{
	int i, j;
	unsigned char* charImage;
	char header[100], * ptr;
	ifstream ifp;

	ifp.open(fname, ios::in | ios::binary);

	if (!ifp)
	{
		cout << "Can't read image: " << fname << endl;
		exit(1);
	}

	// read header

	type = false; // PGM

	ifp.getline(header, 100, '\n');
	if ((header[0] == 80) && (header[1] == 53))
	{
		type = false;
	}
	else if ((header[0] == 80) && (header[1] == 54))
	{
		type = true;
	}
	else
	{
		cout << "Image " << fname << " is not PGM or PPM" << endl;
		exit(1);
	}

	ifp.getline(header, 100, '\n');
	while (header[0] == '#')
		ifp.getline(header, 100, '\n');

	M = strtol(header, &ptr, 0);
	N = atoi(ptr);

	ifp.getline(header, 100, '\n');

	Q = strtol(header, &ptr, 0);

	ifp.close();

	return(1);
}
int writeImage(char fname[], Image& image)
{
	int i, j;
	int N, M, Q;
	unsigned char* charImage;
	ofstream ofp;

	image.getImageInfo(N, M, Q);

	charImage = (unsigned char*) new unsigned char[M * N];
	int val;
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < M; j++)
		{
			val = image.getPixelVal(i, j);
			charImage[i * M + j] = (unsigned char)val;
		}
	}

	ofp.open(fname, ios::out | ios::binary);
	if (!ofp)
	{
		cout << "Can't open file: " << fname << endl;
		exit(1);
	}

	ofp << "P5" << endl;
	ofp << M << " " << N << endl;
	ofp << Q << endl;
	ofp.write(reinterpret_cast<char*>(charImage), (M * N) * sizeof(unsigned char));
	if (ofp.fail())
	{
		cout << "Can't write image " << fname << endl;
		exit(0);
	}
	ofp.close();
	delete[] charImage;
	return(1);
}